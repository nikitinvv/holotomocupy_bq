#include "hip/hip_runtime.h"
#include "kernels_holo.cu"
#include "holo.cuh"
#include<stdio.h>
#define EPS 1e-3

holo::holo(size_t n0e_, size_t n1e_, size_t n0_, size_t n1_, size_t ntheta_) {

  n0e = n0e_; 
  n1e = n1e_;
  n0 = n0_; 
  n1 = n1_;
  ntheta = ntheta_;

  mu0 = -log(EPS) / (2 * n0e * n0e);
  mu1 = -log(EPS) / (2 * n1e * n1e);
  m0 = ceil(2 * n0e * 1 / PI * sqrt(-mu0 * log(EPS) + (mu0 * n0e) * (mu0 * n0e) / 4));
  m1 = ceil(2 * n1e * 1 / PI * sqrt(-mu1 * log(EPS) + (mu1 * n1e) * (mu1 * n1e) / 4));

  int ffts[2];
  int idist;
  int inembed[2];
  // holo 2d
  ffts[0] = 2 * n1e;
  ffts[1] = 2 * n0e;
  idist = (2 * n0e + 2 * m0) * (2 * n1e + 2 * m1);
  inembed[0] = (2 * n1e + 2 * m1);
  inembed[1] = (2 * n0e + 2 * m0);

  hipMalloc((void **)&fdee2d, ntheta*(2 * n1e + 2 * m1) * (2 * n0e + 2 * m0) * sizeof(float2));

  hipfftPlanMany(&plan2dchunk, 2, ffts, inembed, 1, idist, inembed, 1, idist, HIPFFT_C2C, ntheta);

  BS2d = dim3(32, 32, 1);
  GS2d0 = dim3(ceil(n0e / (float)BS2d.x), ceil(n1e / (float)BS2d.y), ceil(ntheta / (float)BS2d.z));
  GS2d1 = dim3(ceil((2 * n0e + 2 * m0) / (float)BS2d.x), ceil((2 * n1e + 2 * m1) / (float)BS2d.y), ceil(ntheta / (float)BS2d.z));
  GS2d2 = dim3(ceil(n0 / (float)BS2d.x), ceil(n1 / (float)BS2d.y), ceil(ntheta / (float)BS2d.z));
 
}

// destructor, memory deallocation
holo::~holo() { free(); }

void holo::free() {
  if (!is_free) {
    hipFree(fdee2d);
    hipfftDestroy(plan2dchunk);
    is_free = true;
  }
}

void holo::fwd_usfft(size_t g_, size_t f_, size_t x_, size_t y_, size_t stream_) {

  f = (float2 *)f_;
  g = (float2 *)g_;
  x = (float *)x_;
  y = (float *)y_;
  stream = (hipStream_t)stream_;    
  
  hipfftSetStream(plan2dchunk, stream);
  hipMemsetAsync(fdee2d, 0, ntheta * (2 * n1e + 2 * m1) * (2 * n0e + 2 * m0) * sizeof(float2),stream);
  
  divker2d<<<GS2d0, BS2d, 0,stream>>>(fdee2d, f, n0e, n1e, ntheta, m0, m1, mu0, mu1, 0);
  fftshiftc2d<<<GS2d1, BS2d, 0,stream>>>(fdee2d, (2 * n0e + 2 * m0), (2 * n1e + 2 * m1), ntheta);
  hipfftExecC2C(plan2dchunk, (hipfftComplex *)&fdee2d[m0 + m1 * (2 * n0e + 2 * m0)].x, (hipfftComplex *)&fdee2d[m0 + m1 * (2 * n0e + 2 * m0)].x, HIPFFT_FORWARD);
  fftshiftc2d<<<GS2d1, BS2d, 0,stream>>>(fdee2d, (2 * n0e + 2 * m0), (2 * n1e + 2 * m1), ntheta);
  wrap2d<<<GS2d1, BS2d, 0,stream>>>(fdee2d, n0e, n1e, ntheta, m0, m1, 0);
  gather2d<<<GS2d2, BS2d, 0,stream>>>(g, fdee2d, x, y, m0, m1, mu0, mu1, n0e, n1e, ntheta, n0, n1, 0);  
}

void holo::adj_usfft(size_t f_, size_t g_, size_t x_, size_t y_, size_t stream_) {

  f = (float2 *)f_;
  g = (float2 *)g_;
  x = (float *)x_;
  y = (float *)y_;
  stream = (hipStream_t)stream_;    
  
  hipfftSetStream(plan2dchunk, stream);

  hipMemsetAsync(fdee2d, 0, ntheta * (2 * n1e + 2 * m1) * (2 * n0e + 2 * m0) * sizeof(float2),stream);
  gather2d<<<GS2d2, BS2d, 0,stream>>>(g, fdee2d, x, y, m0, m1, mu0, mu1, n0e, n1e,ntheta, n0, n1, 1);  
  wrap2d<<<GS2d1, BS2d, 0,stream>>>(fdee2d, n0e, n1e, ntheta, m0, m1, 1);
  fftshiftc2d<<<GS2d1, BS2d, 0,stream>>>(fdee2d, (2 * n0e + 2 * m0), (2 * n1e + 2 * m1), ntheta);
  hipfftExecC2C(plan2dchunk, (hipfftComplex *)&fdee2d[m0 + m1 * (2 * n0e + 2 * m0)].x, (hipfftComplex *)&fdee2d[m0 + m1 * (2 * n0e + 2 * m0)].x, HIPFFT_BACKWARD);
  fftshiftc2d<<<GS2d1, BS2d, 0,stream>>>(fdee2d, (2 * n0e + 2 * m0), (2 * n1e + 2 * m1), ntheta);
  divker2d<<<GS2d0, BS2d, 0,stream>>>(fdee2d, f, n0e, n1e, ntheta, m0, m1, mu0, mu1, 1);
}

void holo::fwd_padsym(size_t g_, size_t f_, size_t pad_width, size_t ns, size_t stream_)
{
  f = (float2 *)f_;
  g = (float2 *)g_;
  stream = (hipStream_t)stream_;    
  
  
  dim3 GS = dim3(ceil((n0+2*pad_width) / (float)BS2d.x), ceil((n1+2*pad_width) / (float)BS2d.y), ceil(ns / (float)BS2d.z));
  pad_sym <<<GS, BS2d, 0,stream>>> (g,f,pad_width,n0,n1,ns,0);  
}

void holo::adj_padsym(size_t g_, size_t f_, size_t pad_width, size_t ns,  size_t stream_)
{
  f = (float2 *)f_;
  g = (float2 *)g_;
  stream = (hipStream_t)stream_;    
  
  dim3 GS = dim3(ceil((n0+2*pad_width) / (float)BS2d.x), ceil((n1+2*pad_width) / (float)BS2d.y), ceil(ns / (float)BS2d.z));
  pad_sym <<<GS, BS2d, 0,stream>>> (f,g,pad_width,n0,n1,ns,1);  
}